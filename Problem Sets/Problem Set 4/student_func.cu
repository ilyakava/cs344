#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>

/* Red Eye Removal
   ===============

   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

__global__
void check_bit(unsigned int* const d_inputVals, unsigned int* const d_outputPredicate,
               const unsigned int bit, const size_t numElems)
{
  // this predicate returns TRUE when the significant bit is not present
  const unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= numElems)
    return;

  int predicate = ((d_inputVals[id] & bit) == 0);
  d_outputPredicate[id] = predicate;
}

__global__
void flip_bit(unsigned int* const d_list, const size_t numElems)
{
  const unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= numElems)
    return;

  d_list[id] = ((d_list[id] + 1) % 2);
}

__global__
void exclusive_blelloch_scan(unsigned int* const d_list, const size_t numElems)
{
  const unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= numElems)
    return;

  // reduce
  unsigned int i;
  for (i = 2; i <= numElems/2; i <<= 1) {
    if ((id + 1) % i == 0) {
      unsigned int neighbor_offset = i>>1;
      d_list[id] += d_list[id - neighbor_offset];
    }
    __syncthreads();
  }
  i >>= 1; // return i to last value before for loop exited
  // reset last to identity element
  if (id == (numElems-1))
    d_list[id] = 0;
  // downsweep
  for (i = i; i >= 2; i >>= 1) {
    if((id + 1) % i == 0) {
      unsigned int neighbor_offset = i>>1;
      unsigned int old_neighbor = d_list[id - neighbor_offset];
      d_list[id - neighbor_offset] = d_list[id]; // copy
      d_list[id] += old_neighbor;
    }
    __syncthreads();
  }
}

__global__
void scatter(unsigned int* const d_input, unsigned int* const d_output,
             unsigned int* const d_predicateTrueScan, unsigned int* const d_predicateFalseScan,
             unsigned int* const d_predicateFalse, unsigned int* const d_numPredicateTrueElements,
             const size_t numElems)
{
  const unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= numElems)
    return;

  unsigned int newLoc;
  if (d_predicateFalse[id] == 1)
    printf("d_numPredicateTrueElements: %i\n", *d_numPredicateTrueElements);
    newLoc = d_predicateFalseScan[id] + *d_numPredicateTrueElements;
  else
    newLoc = d_predicateTrueScan[id];
  assert(newLoc < numElems);
  d_output[newLoc] = d_input[id];
}

unsigned int* d_predicate;
unsigned int* d_predicateTrueScan;
unsigned int* d_predicateFalseScan;
unsigned int* d_numPredicateTrueElements;

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
  printf("numElems: %i\n", numElems);

  size_t size = sizeof(unsigned int) * numElems;
  int blockSize = 1024;
  int gridSize = 1 + (numElems / blockSize);

  unsigned int h_predicateTrue[numElems];
  unsigned int h_predicateTrueScan[numElems];
  unsigned int nsb;
  unsigned int* h_numPredicateTrueElements = (unsigned int *)malloc(sizeof(unsigned int));

  checkCudaErrors(hipMalloc((void**)&d_predicate, size));
  checkCudaErrors(hipMalloc((void**)&d_predicateTrueScan, size));
  checkCudaErrors(hipMalloc((void**)&d_predicateFalseScan, size));
  checkCudaErrors(hipMalloc((void**)&d_numPredicateTrueElements, sizeof(unsigned int)));

  unsigned int max_bits = 32;
  for (unsigned int bit = 0; bit < max_bits; bit++) {
    nsb = 1<<bit;
    // create predicateTrue
    if ((bit + 1) % 2 == 1) {
      check_bit<<<gridSize, blockSize>>>(d_inputVals, d_predicate, nsb, numElems);
    } else {
      check_bit<<<gridSize, blockSize>>>(d_outputVals, d_predicate, nsb, numElems);
    }
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    // scan predicateTrue
    checkCudaErrors(hipMemcpy(d_predicateTrueScan, d_predicate, size, hipMemcpyDeviceToDevice));
    exclusive_blelloch_scan<<<gridSize, blockSize>>>(d_predicateTrueScan, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    // determine offset of 2nd bin, i.e. how many items are in the 1st bin,
    // i.e. for how many the predicate is TRUE
    checkCudaErrors(hipMemcpy(&h_predicateTrue, d_predicate,
                               sizeof(unsigned int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&h_predicateTrueScan, d_predicateTrueScan,
                               sizeof(unsigned int), hipMemcpyDeviceToHost));
    *h_numPredicateTrueElements = h_predicateTrueScan[numElems-1] + h_predicateTrue[numElems-1];
    printf("nsb: %i h_numPredicateTrueElements: %i\n", nsb, *h_numPredicateTrueElements);
    checkCudaErrors(hipMemcpy(d_numPredicateTrueElements, h_numPredicateTrueElements,
                               sizeof(unsigned int), hipMemcpyHostToDevice));
    // transform predicateTrue -> predicateFalse
    flip_bit<<<gridSize, blockSize>>>(d_predicate, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    // scan predicateFalse
    checkCudaErrors(hipMemcpy(d_predicateFalseScan, d_predicate, size, hipMemcpyDeviceToDevice));
    exclusive_blelloch_scan<<<gridSize, blockSize>>>(d_predicateFalseScan, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    // scatter values (flip input/output depending on iteration)
    if ((bit + 1) % 2 == 1) {
      scatter<<<gridSize, blockSize>>>(d_inputVals, d_outputVals, d_predicateTrueScan, d_predicateFalseScan,
                                       d_predicate, d_numPredicateTrueElements, numElems);
      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
      // scatter<<<gridSize, blockSize>>>(d_inputPos, d_outputPos, d_predicateTrueScan, d_predicateFalseScan,
      //                                  d_predicate, d_numPredicateTrueElements, numElems);
      // hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    } else {
      scatter<<<gridSize, blockSize>>>(d_outputVals, d_inputVals, d_predicateTrueScan, d_predicateFalseScan,
                                       d_predicate, d_numPredicateTrueElements, numElems);
      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
      // scatter<<<gridSize, blockSize>>>(d_outputPos, d_inputPos, d_predicateTrueScan, d_predicateFalseScan,
      //                                  d_predicate, d_numPredicateTrueElements, numElems);
      // hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    }
  }
  checkCudaErrors(hipFree(d_predicate));
  checkCudaErrors(hipFree(d_predicateTrueScan));
  checkCudaErrors(hipFree(d_predicateFalseScan));
  checkCudaErrors(hipFree(d_numPredicateTrueElements));
}

#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>

/* Red Eye Removal
   ===============

   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

__global__
void check_bit(unsigned int* const d_inputVals, unsigned int* const d_outputPredicate,
               const unsigned int bit, const size_t numElems)
{
  // this predicate returns TRUE when the significant bit is not present
  const unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= numElems)
    return;

  int predicate = ((d_inputVals[id] & bit) == 0);
  d_outputPredicate[id] = predicate;
}

__global__
void flip_bit(unsigned int* const d_list, const size_t numElems)
{
  const unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= numElems)
    return;

  d_list[id] = ((d_list[id] + 1) % 2);
}

__global__
void exclusive_blelloch_scan(unsigned int* const d_list, const size_t numElems)
{
  const unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= numElems)
    return;

  // reduce
  unsigned int i;
  for (i = 2; i <= numElems; i <<= 1) {
    if ((id + 1) % i == 0) {
      unsigned int neighbor_offset = i>>1;
      d_list[id] += d_list[id - neighbor_offset];
    }
    __syncthreads();
  }
  i >>= 1; // return i to last value before for loop exited
  // reset last to identity element
  if (id == (numElems-1))
    d_list[id] = 0;

  // __syncthreads();
  // printf("in kernel: %i %i %i %i %i %i %i %i\n", d_list[0], d_list[1], d_list[2], d_list[3], d_list[4], d_list[5], d_list[6], d_list[7]);

  // downsweep
  for (i = i; i >= 2; i >>= 1) {
    // printf("downsweep: %i\n", i);
    __syncthreads();
    if((id + 1) % i == 0) {
      unsigned int neighbor_offset = i>>1;
      unsigned int old_neighbor = d_list[id - neighbor_offset];
      d_list[id - neighbor_offset] = d_list[id]; // copy
      d_list[id] += old_neighbor;
    }
  }
}

__global__
void scatter(unsigned int* const d_input, unsigned int* const d_output,
             unsigned int* const d_predicateTrueScan, unsigned int* const d_predicateFalseScan,
             unsigned int* const d_predicateFalse, unsigned int* const d_numPredicateTrueElements,
             const size_t numElems)
{
  const unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= numElems)
    return;

  unsigned int newLoc;
  if (d_predicateFalse[id] == 1) {
    newLoc = d_predicateFalseScan[id] + *d_numPredicateTrueElements;
  } else {
    newLoc = d_predicateTrueScan[id];
  }


  if (newLoc >= numElems)
    printf("ALERT d_predicateFalse[id]: %i newLoc: %i numElems: %i\n", d_predicateFalse[id], newLoc, numElems);


  d_output[newLoc] = d_input[id];
}

unsigned int* d_predicate;
unsigned int* d_predicateTrueScan;
unsigned int* d_predicateFalseScan;
unsigned int* d_numPredicateTrueElements;

// DEBUG
void print_array(unsigned int* array, size_t length)
{
  for (int i = 0; i < length; i++)
    printf("%i ", array[i]);
  printf("\n");
}


void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
  printf("numElems: %i\n", numElems);


  // DEBUG
  size_t myNumElems = 17;


  size_t size = sizeof(unsigned int) * myNumElems;
  int blockSize = 1024;
  int gridSize = 1 + (myNumElems / blockSize);

  unsigned int h_array[myNumElems];

  unsigned int h_predicateTrue[myNumElems];
  unsigned int h_predicateTrueScan[myNumElems];
  unsigned int nsb;
  unsigned int* h_numPredicateTrueElements = (unsigned int *)malloc(sizeof(unsigned int));

  checkCudaErrors(hipMalloc((void**)&d_predicate, size));
  checkCudaErrors(hipMalloc((void**)&d_predicateTrueScan, size));
  checkCudaErrors(hipMalloc((void**)&d_predicateFalseScan, size));
  checkCudaErrors(hipMalloc((void**)&d_numPredicateTrueElements, sizeof(unsigned int)));

  unsigned int max_bits = 30;
  for (unsigned int bit = 0; bit < max_bits; bit++) {


    // DEBUG
    // checkCudaErrors(hipMemcpy(&h_array, d_inputVals, size, hipMemcpyDeviceToHost));
    // printf("bit %i array:\n", bit);
    // print_array(h_array, myNumElems);


    nsb = 1<<bit;
    // create predicateTrue
    if ((bit + 1) % 2 == 1) {
      check_bit<<<gridSize, blockSize>>>(d_inputVals, d_predicate, nsb, myNumElems);
    } else {
      check_bit<<<gridSize, blockSize>>>(d_outputVals, d_predicate, nsb, myNumElems);
    }
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    // scan predicateTrue
    checkCudaErrors(hipMemcpy(d_predicateTrueScan, d_predicate, size, hipMemcpyDeviceToDevice));
    exclusive_blelloch_scan<<<gridSize, blockSize>>>(d_predicateTrueScan, myNumElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    // determine offset of 2nd bin, i.e. how many items are in the 1st bin,
    // i.e. for how many the predicate is TRUE
    checkCudaErrors(hipMemcpy(&h_predicateTrue, d_predicate,
                               size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&h_predicateTrueScan, d_predicateTrueScan,
                               size, hipMemcpyDeviceToHost));
    *h_numPredicateTrueElements = h_predicateTrueScan[myNumElems-1] + h_predicateTrue[myNumElems-1];
    // printf("h_numPredicateTrueElements: %i\n", *h_numPredicateTrueElements);
    checkCudaErrors(hipMemcpy(d_numPredicateTrueElements, h_numPredicateTrueElements,
                               sizeof(unsigned int), hipMemcpyHostToDevice));


    // DEBUG
    // printf("h_predicateTrue:\n");
    // print_array(h_predicateTrue, myNumElems);
    // printf("h_predicateTrueScan:\n");
    // print_array(h_predicateTrueScan, myNumElems);
    // printf("h_numPredicateTrueElements: %i\n", *h_numPredicateTrueElements);


    // transform predicateTrue -> predicateFalse
    flip_bit<<<gridSize, blockSize>>>(d_predicate, myNumElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    // scan predicateFalse
    checkCudaErrors(hipMemcpy(d_predicateFalseScan, d_predicate, size, hipMemcpyDeviceToDevice));
    exclusive_blelloch_scan<<<gridSize, blockSize>>>(d_predicateFalseScan, myNumElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    // scatter values (flip input/output depending on iteration)
    if ((bit + 1) % 2 == 1) {
      scatter<<<gridSize, blockSize>>>(d_inputVals, d_outputVals, d_predicateTrueScan, d_predicateFalseScan,
                                       d_predicate, d_numPredicateTrueElements, myNumElems);
      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
      // scatter<<<gridSize, blockSize>>>(d_inputPos, d_outputPos, d_predicateTrueScan, d_predicateFalseScan,
      //                                  d_predicate, d_numPredicateTrueElements, myNumElems);
      // hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    } else {
      scatter<<<gridSize, blockSize>>>(d_outputVals, d_inputVals, d_predicateTrueScan, d_predicateFalseScan,
                                       d_predicate, d_numPredicateTrueElements, myNumElems);
      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
      // scatter<<<gridSize, blockSize>>>(d_outputPos, d_inputPos, d_predicateTrueScan, d_predicateFalseScan,
      //                                  d_predicate, d_numPredicateTrueElements, myNumElems);
      // hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    }
  }
  checkCudaErrors(hipFree(d_predicate));
  checkCudaErrors(hipFree(d_predicateTrueScan));
  checkCudaErrors(hipFree(d_predicateFalseScan));
  checkCudaErrors(hipFree(d_numPredicateTrueElements));
}
